#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_math.h>
#include "random.h"
#include "helper.h"
using namespace optix;

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, tangent, attribute tangent, );
rtDeclareVariable(float3, bigTangent, attribute bigTangent, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(int, radiance_ray_type, , );

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_color, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(InfoRay_mask, prd_maskInfo, rtPayload, );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(float3, reflectivity, , );

rtDeclareVariable(int, useNormalMap, , );


rtDeclareVariable(float3, Kcrq, , );
rtDeclareVariable(int, reflectionNums, , );

rtTextureSampler<float4, 2> normal_map;
rtTextureSampler<float4, 2> normal_map1;

rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float, y_angle, , );
rtDeclareVariable(float, x_angle, , );
rtDeclareVariable(unsigned int, frame_number, , );

rtDeclareVariable(int, shading_ray, , );
rtDeclareVariable(int, shadow_ray, , );
rtDeclareVariable(int, refract_Ray, , );
rtDeclareVariable(int, light_distri_ray, , );

rtDeclareVariable(float, exponent, , );
rtDeclareVariable(float3, emission_color, , );

RT_PROGRAM void glossy_bsdf()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);

	float t = dot(-ray.direction, world_geo_normal);
	float3 color = Ka * ambient_color;
	float3 hit_point = ray.origin + t_hit * ray.direction;

	float3 result = make_float3(1.0f);

	if (t > 0){
		float2 sample;
		sample.x = rnd(prd_radiance.seed);
		sample.y = rnd(prd_radiance.seed);

		float bsdf_pdf;
		float bsdf_val;
		float3 dir_r = reflect(ray.direction, ffnormal);

		float3 v1, v2;
		createONB(dir_r, v1, v2);
		float3 direction = sample_phong_lobe(sample, exponent, v1, v2,dir_r, bsdf_pdf, bsdf_val);
		direction = normalize(direction);

		PerRayData_radiance prd;
		prd.radiance = make_float3(0);
		prd.attenuation = make_float3(1);
		prd.result = make_float3(0);
		prd.depth = 0;

		optix::Ray ray(hit_point, direction,
			radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

		rtTrace(top_object, ray, prd);

		if (bsdf_pdf > 0)
		{
			/*prd.attenuation = prd.attenuation * bsdf_val / bsdf_pdf
				*fabs(dot(direction, ffnormal));*/

			prd.attenuation = prd.attenuation * bsdf_val / bsdf_pdf;
		}

		prd_radiance.radiance = prd.attenuation * prd.radiance;
	}
}

RT_PROGRAM
void diffuseEmetter(){
	prd_radiance.radiance = emission_color;
	prd_radiance.done = true;
}

rtDeclareVariable(float3, point_light_pos, , );
RT_PROGRAM void phong()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float kd = 0.3;
	float ks = 0.7;
	
	if (dot(-ray.direction, world_shading_normal) > 0){
		float3 hitpoint = ray.origin + t_hit * ray.direction;
		float3 dir_r = reflect(ray.direction, ffnormal);

		float3 white = make_float3(1);
		float diffuse = kd * dot(normalize(point_light_pos - hitpoint), ffnormal);

		float3 halfV = point_light_pos - hitpoint + ray.direction;
		halfV = normalize(halfV);
		float specular = ks * powf(dot(halfV, ffnormal), exponent);

		prd_radiance.radiance = white *(specular + diffuse);
	}
}



RT_PROGRAM void color(){
	prd_radiance.radiance = make_float3(1,1,0);
}



RT_PROGRAM void shadow()
{
	prd_radiance.attenuation = make_float3(0);
}