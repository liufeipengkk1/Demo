#include "hip/hip_runtime.h"



#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_math_namespace.h>
#include "RayInfoStruct.h"
#include "random.h"
#include "helper.h"
using namespace optix;

//variable
rtDeclareVariable(int, radiance_ray_type, , );
rtDeclareVariable(int, shadow_ray_type, , );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, bd_color, , );
rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, ambient_color, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, frame_number, , );

rtBuffer<float4, 2>  output_buffer;
rtBuffer<float4, 2> hdr_buffer;

//sematic variable
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texture_coord, attribute texture_coord, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


//we also need to know current ray payload for ray travesal information
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

//���˥��ϵ��
rtDeclareVariable(float3, Kcrq, , );

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	float2 pixel = make_float2(launch_index) / make_float2(screen) * 2 - 1;
	
	float3 pixel_color;

	//jitter
	float2 dt = 1.0 / make_float2(screen) * 2;

	unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;
	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);
	
	PerRayData_radiance prd;
	//init prd 
	prd.depth = 0;
	prd.result = make_float3(0, 0, 0);
	prd.radiance = make_float3(0, 0, 0);
	prd.attenuation = make_float3(1.f);
	prd.countEmitted = true;
	prd.done = false;
	prd.seed = seed;

	//�������1 һ������ �������������������Ȼ���и����Ե����
	if (sqrt_num_samples == 1){
		float2 d = pixel;

		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x * U + d.y*V + W);
		optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

		rtTrace(top_object, ray, prd);

		prd.result = prd.radiance;
	}
	else{
		//һ������ȡsamples_per_pixel������
		for (int y = 0; y < sqrt_num_samples; y++){
			for (int x = 0; x < sqrt_num_samples; x++){

				//use jitter for random ray ... 
				float2 r = make_float2(x - rnd(seed), y - rnd(seed)) * dt;


				float2 d = pixel + r;

				float3 ray_origin = eye;
				float3 ray_direction = normalize(d.x * U + d.y*V + W);
				optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

				rtTrace(top_object, ray, prd);

				prd.result += prd.radiance;
			}
		}
	}
	

	
	prd.result /= samples_per_pixel;
	
	pixel_color = prd.radiance;
	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float b = 1 - a;
		float3 old_color = make_float3(hdr_buffer[launch_index]);

		pixel_color = a * pixel_color + b * old_color;
		hdr_buffer[launch_index] = make_float4(pixel_color, 1);
	}
	else
	{
		output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
		hdr_buffer[launch_index] = make_float4(pixel_color, 1.0f);
	}
	
	output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
}

RT_PROGRAM void miss()
{
	prd_radiance.radiance = bg_color;
	prd_radiance.done = true;
}


RT_PROGRAM void exception(){
	output_buffer[launch_index] = make_float4(bg_color, 1);
}

rtDeclareVariable(float3, color, , );

RT_PROGRAM void closest_hit_program()
{	
	prd_radiance.radiance = color ;
}

