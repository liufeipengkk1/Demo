#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "cudaUtil.h"
#include "../helper.h"
#include "../random.h"

using namespace optix;

//global
rtDeclareVariable(rtObject, top_Object, , );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(ShaderRay, prd_radiance, rtPayload, );

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtTextureSampler<float4, 2> normalMap;

RT_PROGRAM void fragment()
{
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 T = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 B = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 texNormal = make_float3(tex2D(normalMap, texcoord.x, texcoord.y));
	texNormal = 2 * texNormal - 1;
	prd_radiance.m_depth += 1;
	if (prd_radiance.m_depth < 7 && !prd_radiance.m_done) {
		//if (dot(-ray.direction, geometric_normal) > 0) {
			float3 hit_point = ray.origin + t_hit * ray.direction;
			float3 R = reflect(ray.direction, texNormal);
			ShaderRay refl_prd;
			refl_prd.m_radiance = make_float3(0, 0, 0);
			refl_prd.m_depth = prd_radiance.m_depth;
			optix::Ray refl_ray(hit_point, R, SHADING_RAY, SCENE_EPSILON);
			rtTrace(top_Object, refl_ray, refl_prd);
			prd_radiance.m_radiance = refl_prd.m_radiance;
		//}
	}
}


rtDeclareVariable(float3, pointLightPos, , );
rtDeclareVariable(float, pointLightRadius, ,);
RT_PROGRAM void glossy_BRDF() {
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 hit_point = ray.origin + t_hit * ray.direction;

	prd_radiance.m_origin = hit_point;

	float2 sample;
	sample.x = rnd(prd_radiance.m_seed);
	sample.y = rnd(prd_radiance.m_seed);
	float bsdf_pdf;
	float bsdf_val;
	float3 dir_out = -ray.direction;
	float3 texNormal = make_float3(tex2D(normalMap, texcoord.x, texcoord.y));
	float3 dir_r = reflect(ray.direction, texNormal);
	float3 v1, v2;
	createONB(dir_r, v1, v2);
	prd_radiance.m_direction = sample_phong_lobe(sample, 10000, v1, v2, dir_r, bsdf_pdf, bsdf_val);  //������ݺ���������ʲô  ����ô�Ĺؼ���   ���ա�
	if (bsdf_val > 0) {
		//�������� 
		float3 d = normalize(-ray.direction) + normalize(prd_radiance.m_direction);
		d = normalize(d);
		float rf0 = 0.15;
		float cosAlpha = dot(d, texNormal);
		float fre = 0;
		if (cosAlpha > 0) {
			fre = rf0 + (1 - rf0)* pow(1 - cosAlpha, 5);
			prd_radiance.m_attenuation = make_float3(bsdf_val / bsdf_pdf);
		}

		prd_radiance.m_attenuation =  make_float3(bsdf_val / bsdf_pdf);
	}
}

RT_PROGRAM void microfacet_BSDF(){

}

rtDeclareVariable(float3, col, ,);
RT_PROGRAM void color()
{
	prd_radiance.m_radiance = col;
	prd_radiance.m_done = true;
}
