#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "cudaUtil.h"

using namespace optix;

rtDeclareVariable(float3, color, , );

rtDeclareVariable(ShaderRay, prd_radiance, rtPayload, );

RT_PROGRAM void fragment()
{
	prd_radiance.m_color = make_float4(color, 1.0);
	prd_radiance.m_done = true;
}
