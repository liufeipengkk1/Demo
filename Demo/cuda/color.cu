#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "cudaUtil.h"

using namespace optix;

rtDeclareVariable(ShaderRay, prd_radiance, rtPayload, );

RT_PROGRAM void fragment()
{
	prd_radiance.m_color = make_float4(1, 0, 1, 1.0);
	prd_radiance.m_done = true;
}
