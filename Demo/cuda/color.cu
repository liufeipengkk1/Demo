#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "cudaUtil.h"
#include "../helper.h"

using namespace optix;

//global
rtDeclareVariable(rtObject, top_Object, , );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(ShaderRay, prd_radiance, rtPayload, );

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void fragment()
{
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	prd_radiance.m_depth += 1;
	if (prd_radiance.m_depth < 7 && !prd_radiance.m_done) {
		//if (dot(-ray.direction, geometric_normal) > 0) {
			float3 hit_point = ray.origin + t_hit * ray.direction;
			float3 R = reflect(ray.direction, world_shade_normal);
			ShaderRay refl_prd;
			refl_prd.m_color = make_float4(0, 0, 0, 0);
			refl_prd.m_depth = prd_radiance.m_depth;
			optix::Ray refl_ray(hit_point, R, SHADING_RAY, SCENE_EPSILON);
			rtTrace(top_Object, refl_ray, refl_prd);
			prd_radiance.m_color = refl_prd.m_color;
		//}
	}
}

rtDeclareVariable(float3, col, ,);
RT_PROGRAM void color()
{
	prd_radiance.m_color = make_float4(col, 1);
	prd_radiance.m_depth += 1;
	prd_radiance.m_done = true;
}
