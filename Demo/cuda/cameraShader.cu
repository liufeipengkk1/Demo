#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "../helper.h"
#include "../random.h"
#include "cudaUtil.h"

using namespace optix;

rtDeclareVariable(float3, eye, ,);
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, w, , );

//global
rtDeclareVariable(rtObject, top_Object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(ShaderRay, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );

rtBuffer<float4, 2>    output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*u + d.y*v + w);

	optix::Ray ray(ray_origin, ray_direction, SHADING_RAY, SCENE_EPSILON);
	
	ShaderRay prd;
	prd.m_color = make_float4(0, 0, 0, 0);
	prd.m_depth = 0;
	prd.m_done = false;

	rtTrace(top_Object, ray, prd);
	
	output_buffer[launch_index] = prd.m_color;
}

rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(float, Y_log_av, , );
rtDeclareVariable(float, Y_max, , );
rtBuffer<float3, 2>  hdr_buffer;
////�����camera
RT_PROGRAM void jitter_pinhole_camera(){
	size_t2 screen = output_buffer.size();
	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;
	unsigned int sqrt_num_samples = 2;//2X����
	float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
	float3 result = make_float3(0.0f);
	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);
	do{
		unsigned int x = samples_per_pixel%sqrt_num_samples;
		unsigned int y = samples_per_pixel / sqrt_num_samples;
		float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
		float2 d = pixel + jitter*jitter_scale;
		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x*u + d.y*v + w);

		ShaderRay prd;
		prd.m_color = make_float4(0, 0, 0, 0);
		prd.m_result = make_float3(0);
		prd.m_radiance = make_float3(0);
		prd.m_depth = 0;
		prd.m_done = false;
		prd.m_seed = seed;
		prd.m_attenuation = make_float3(1.f);

		for (;;) {
			optix::Ray ray(ray_origin, ray_direction, SHADING_RAY, SCENE_EPSILON, RT_DEFAULT_MAX);
			rtTrace(top_Object, ray, prd);

			if (prd.m_done) {
				prd.m_result += prd.m_radiance * prd.m_attenuation;
				break;
			}

			if (prd.m_depth >= 2){
				float pcont = fmaxf(prd.m_attenuation);
				if (rnd(prd.m_seed) >= pcont)
					break;
				prd.m_attenuation /= pcont;
			}
			prd.m_depth++;
			prd.m_result += prd.m_radiance * prd.m_attenuation;
			ray_origin = prd.m_origin;
			ray_direction = prd.m_direction;
		} 

		result += prd.m_result;
		seed = prd.m_seed;
	} while (--samples_per_pixel);

	result = result / (sqrt_num_samples*sqrt_num_samples);
	float3 pixel_color = make_float3(result.x, result.y, result.z);
	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float b = ((float)frame_number - 1.0f) * a;
		hdr_buffer[launch_index] = a * pixel_color + b * hdr_buffer[launch_index];
	}
	else
		hdr_buffer[launch_index] = pixel_color;

	//need toemapping for hdr? 
	//output_buffer[launch_index] = make_float4(tonemap(hdr_buffer[launch_index], Y_log_av, Y_max), 1.0f);
	output_buffer[launch_index] = make_float4(hdr_buffer[launch_index], 1.0);
}

//rtTextureSampler<float4, 2> envmap;
rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(float3, bgColor,,);
rtDeclareVariable(float, intensity, , );
RT_PROGRAM void backGround()
{
	/*float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));*/

	float x = ray.direction.x;
	float y = ray.direction.y;
	float z = ray.direction.z;
	float p = acosf(y);
	float h = atan2(x, z);
	float pi = 3.14159265358979323846;
	if (x > 0)
		h = h;
	else
		h = 2 * pi + h;
	
	float v = p / pi;
	float u = h / pi * 0.5;
	prd_radiance.m_radiance = make_float3(tex2D(envmap, u, v));
	//prd_radiance.m_radiance = make_float3(0);
	prd_radiance.m_done = true;
	//prd_radiance.m_color = make_float4(0,1,0, 1.0);
}

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(1, 1, 0, 1);
}