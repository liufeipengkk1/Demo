#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "../helper.h"
#include "cudaUtil.h"

using namespace optix;

rtDeclareVariable(float3, eye, ,);
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, w, , );

//global
rtDeclareVariable(rtObject, top_Object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(ShaderRay, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );

rtBuffer<float4, 2>    output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*u + d.y*v + w);

	optix::Ray ray(ray_origin, ray_direction, SHADING_RAY, SCENE_EPSILON);
	
	ShaderRay prd;
	prd.m_depth = 0;
	prd.m_done = false;

	rtTrace(top_Object, ray, prd);
	
	output_buffer[launch_index] = prd.m_color;
}

rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(float3, bgColor,,);
RT_PROGRAM void backGround()
{
	//float theta = atan2f(ray.direction.x, ray.direction.z);
	//float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	//float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	//float v = 0.5f * (1.0f + sin(phi));
	//prd_radiance.m_color = tex2D(envmap, u, v);

	prd_radiance.m_color = make_float4(0,1,0, 1.0);
}

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(1, 1, 0, 1);
}