#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_math_namespace.h>
#include "RayInfoStruct.h"
#include "random.h"
using namespace optix;


rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );


///3 types of ray.
rtDeclareVariable(int, shading_ray, , );
rtDeclareVariable(int, shadow_ray, , );
rtDeclareVariable(int, light_distri_ray, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtBuffer<float4, 2>  output_buffer;


RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	float2 pixel = make_float2(launch_index) / make_float2(screen) * 2 - 1;
	float3 pixel_color;

	//jitter
	float2 dt = 1.0 / make_float2(screen) * 2;

	unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;
	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

	PerRayData_radiance prd;
	//init prd 
	prd.depth = 0;
	prd.result = make_float3(0, 0, 0);
	prd.radiance = make_float3(0, 0, 0);
	prd.attenuation = make_float3(1.f);
	prd.countEmitted = true;
	prd.done = false;
	prd.seed = seed;

	//�������1 һ������ �������������������Ȼ���и����Ե����
	if (sqrt_num_samples == 1){
		float2 d = pixel;

		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x * U + d.y*V + W);
		optix::Ray ray(ray_origin, ray_direction, shading_ray, scene_epsilon);

		rtTrace(top_object, ray, prd);

		prd.result += prd.radiance;
	}
	else{
		//һ������ȡsamples_per_pixel������
		for (int y = 0; y < sqrt_num_samples; y++){
			for (int x = 0; x < sqrt_num_samples; x++){

				//use jitter for random ray ... 
				float2 r = make_float2(x - rnd(seed), y - rnd(seed)) * dt;
				float2 d = pixel + r;
				float3 ray_origin = eye;
				float3 ray_direction = normalize(d.x * U + d.y*V + W);
				optix::Ray ray(ray_origin, ray_direction, shading_ray, scene_epsilon);

				rtTrace(top_object, ray, prd);

				prd.result += prd.radiance;
			}
		}
	}



	prd.result /= samples_per_pixel;

	pixel_color = prd.radiance;

	float a = 1.0f / (float)frame_number;
	float b = 1 - a;
	float3 old_color = make_float3(output_buffer[launch_index]);
	output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 1);
	//output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
}


RT_PROGRAM void miss()
{
	prd_radiance.radiance = bg_color;
}


RT_PROGRAM void exception(){
	output_buffer[launch_index] = make_float4(bg_color, 1);
}
