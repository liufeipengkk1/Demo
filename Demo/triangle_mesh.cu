#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3> vertex_buffer;	//vertex
rtBuffer<float3> normal_buffer; //normal
rtBuffer<float2> texcoord_buffer; //uv
rtBuffer<float3> tangent_buffer;  //tangent
rtBuffer<float3> bigTangent_buffer;  // bigTangent
rtBuffer<int3> tri_indice;

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, tangent, attribute tangent, );
rtDeclareVariable(float3, bigTangent, attribute bigTangent, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect(int primIdx)
{
	const float3 p0 = vertex_buffer[tri_indice[primIdx].x];
	const float3 p1 = vertex_buffer[tri_indice[primIdx].y];
	const float3 p2 = vertex_buffer[tri_indice[primIdx].z];

	const float3 t1 = tangent_buffer[tri_indice[primIdx].x];
	const float3  bt1 = bigTangent_buffer[tri_indice[primIdx].x];
	//intersect ray with triangle
		
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

		if (rtPotentialIntersection(t)) {

			geometric_normal = normalize(n);
			tangent = t1;
			bigTangent = bt1;
			if (normal_buffer.size() == 0) {
				shading_normal = geometric_normal;
			}
			else {
				float3 n0 = normal_buffer[3 * primIdx];
				float3 n1 = normal_buffer[3 * primIdx+1];
				float3 n2 = normal_buffer[3 * primIdx+2];
				shading_normal = normalize(n1*beta + n2*gamma + n0*(1.0f - beta - gamma));
				//shading_normal = geometric_normal;
			}

			if (texcoord_buffer.size() == 0) {
				texcoord = make_float2(0.0f, 0.0f);
			}
			else {
				float2 t0 = texcoord_buffer[3 * primIdx];
				float2 t1 = texcoord_buffer[3 * primIdx+1];
				float2 t2 = texcoord_buffer[3 * primIdx+2];
				texcoord = t1*beta + t2*gamma + t0*(1.0f - beta - gamma);
			}
			rtReportIntersection(0);
		}
	}
}


RT_PROGRAM void mesh_bounds(int primIdx, float result[6])
{
	const float3 v0 = vertex_buffer[tri_indice[primIdx].x];
	const float3 v1 = vertex_buffer[tri_indice[primIdx].y];
	const float3 v2 = vertex_buffer[tri_indice[primIdx].z];

	const float  area = length(cross(v1 - v0, v2 - v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else {
		aabb->invalidate();
	}
}
