#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
//#include "RayInfoStruct.h"

using namespace optix;

rtBuffer<float3> vertex_buffer;	//vertex
rtBuffer<float3> normal_buffer; //normal
rtBuffer<float2> texcoord_buffer; //uv
rtBuffer<float3> tangent_buffer;  //tangent
rtBuffer<float3> bigTangent_buffer;// bigTangent
rtBuffer<int3> tri_indice;

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, tangent, attribute tangent, );
rtDeclareVariable(float3, bigTangent, attribute bigTangent, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect(int primIdx)
{
	const float3 p0 = vertex_buffer[tri_indice[primIdx].x];
	const float3 p1 = vertex_buffer[tri_indice[primIdx].y];
	const float3 p2 = vertex_buffer[tri_indice[primIdx].z];

	const float3 t1 = tangent_buffer[tri_indice[primIdx].x];
	const float3  bt1 = bigTangent_buffer[tri_indice[primIdx].x];
	//intersect ray with triangle
		
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

		if (rtPotentialIntersection(t)) {

			geometric_normal = normalize(n);
			tangent = t1;
			bigTangent = bt1;
			if (normal_buffer.size() == 0) {
				shading_normal = geometric_normal;
			}
			else {
				float3 n0 = normal_buffer[3 * primIdx];
				float3 n1 = normal_buffer[3 * primIdx+1];
				float3 n2 = normal_buffer[3 * primIdx+2];
				//shading_normal = normalize(n1*beta + n2*gamma + n0*(1.0f - beta - gamma));
				shading_normal = geometric_normal;
			}

			if (texcoord_buffer.size() == 0) {
				texcoord = make_float2(0.0f, 0.0f);
			}
			else {
				float2 t0 = texcoord_buffer[3 * primIdx];
				float2 t1 = texcoord_buffer[3 * primIdx+1];
				float2 t2 = texcoord_buffer[3 * primIdx+2];
				texcoord = t1*beta + t2*gamma + t0*(1.0f - beta - gamma);
			}
			rtReportIntersection(0);
		}
	}
	
		/*
	//pom = t1 * p0p1 + t2 * p0p2;
	//==>m = t1*p1 + t2 * p2 + (1 - t1 - t2) * p0;
	//��M�������������� t1 + t2 < 1,t1 t2��С��0 �����t1 ��t2 �������������ڷ�����uv����ͷ�����
	
	float3 e0 = p1 - p0;
	float3 e1 = p2 - p0;

	float3 n = cross(e0, e1);
	float3 origin = ray.origin;
	float3 direction = ray.direction;
	float dt = dot(direction, n);  
	
	//�ཻ
	if (dt != 0){
		float d = 1 / dot(direction, n);
		float t = dot(p0 - origin, n)*d;

		if (t >= ray.tmin && t <= ray.tmax){
			
			//���㽻��
			float3 p = origin + t * direction;
			float3 R = p - p0;
			float3 Q1 = p1 - p0;
			float3 Q2 = p2 - p0;

			//R = t1* Q1 + t2 * Q2
			float Q1Q1 = dot(Q1, Q1);
			float Q2Q2 = dot(Q2, Q2);
			float RQ1 = dot(R, Q1);
			float RQ2 = dot(R, Q2);
			float Q1Q2 = dot(Q1, Q2);

			float dd = 1 / (Q1Q1 * Q2Q2 - Q1Q2*Q1Q2);
			float t1 = (Q2Q2*RQ1 - Q1Q2*RQ2) / dd;
			float t2 = (Q1Q1*RQ2 - Q1Q2*RQ1) / dd;

			if ((t1 + t2) <= 1 && t1 >= 0 && t2 >= 0){
				if (rtPotentialIntersection(t)){

					geometric_normal = normalize(n);
					if (normal_buffer.size() == 0) {
						shading_normal = geometric_normal;
					}
					else {
						float3 n0 = normal_buffer[3 * primIdx];
						float3 n1 = normal_buffer[3 * primIdx + 1];
						float3 n2 = normal_buffer[3 * primIdx + 2];
						shading_normal = normalize(n1*t1 + n2*t2 + n0*(1.0f - t1 - t2));
					}

					if (texcoord_buffer.size() == 0) {
						texcoord = make_float3(0.0f, 0.0f, 0.0f);
					}
					else {
						float2 tex0 = texcoord_buffer[3 * primIdx];
						float2 tex1 = texcoord_buffer[3 * primIdx + 1];
						float2 tex2 = texcoord_buffer[3 * primIdx + 2];
						texcoord = make_float3(tex1*t1 + tex2*t2 + tex0*(1.0f - t1 - t2));
					}

					rtReportIntersection(0);
				}
			}
		}

	}
	*/
	
}


RT_PROGRAM void mesh_bounds(int primIdx, float result[6])
{
	//const int3 v_idx = index_buffer[primIdx];
	/*const float3 v0 = vertex_buffer[v_idx.x];
	const float3 v1 = vertex_buffer[v_idx.y];
	const float3 v2 = vertex_buffer[v_idx.z];*/

	const float3 v0 = vertex_buffer[tri_indice[primIdx].x];
	const float3 v1 = vertex_buffer[tri_indice[primIdx].y];
	const float3 v2 = vertex_buffer[tri_indice[primIdx].z];

	const float  area = length(cross(v1 - v0, v2 - v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else {
		aabb->invalidate();
	}
}
